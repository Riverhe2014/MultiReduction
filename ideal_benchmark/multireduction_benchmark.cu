#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
using namespace std::chrono;

#define _warpSize (32)
#define intsPerVector (32)
#define vectorsPerLoop (32)
#define warpsPerBlock (8)
#define loopsPerWarp (64)
#define vectorsPerWarp (vectorsPerLoop * loopsPerWarp)
#define vectorsPerBlock (vectorsPerWarp * warpsPerBlock)
#define blocksPerSM (8)
#define scale (1)

template<typename T>
__launch_bounds__(_warpSize * warpsPerBlock, blocksPerSM)
__global__ void
add32_naive(const T *g_V, T *g_S)
{
	T v[vectorsPerLoop];
	int readOffset = (blockIdx.x  * intsPerVector * vectorsPerBlock)
				   + (threadIdx.y * intsPerVector * vectorsPerWarp)
				   + (threadIdx.x);
	int writeOffset = (blockIdx.x  * vectorsPerBlock)
	                + (threadIdx.y * vectorsPerWarp)
					+ (threadIdx.x);
	for (int loop = 0; loop < loopsPerWarp; loop++, writeOffset += vectorsPerLoop) {
		#pragma unroll
		for (int i = 0; i < vectorsPerLoop; i++, readOffset += _warpSize)
			v[i] = g_V[readOffset];
		#pragma unroll
		for (int j = 1; j < _warpSize; j <<= 1)
			#pragma unroll
			for (int i = 0; i < vectorsPerLoop; i++)
				v[i] += __shfl_xor(v[i], j);
		{
			if (threadIdx.x == 1) v[0] = v[1];
			if (threadIdx.x == 2) v[0] = v[2];
			if (threadIdx.x == 3) v[0] = v[3];
			if (threadIdx.x == 4) v[0] = v[4];
			if (threadIdx.x == 5) v[0] = v[5];
			if (threadIdx.x == 6) v[0] = v[6];
			if (threadIdx.x == 7) v[0] = v[7];
			if (threadIdx.x == 8) v[0] = v[8];
			if (threadIdx.x == 9) v[0] = v[9];
			if (threadIdx.x == 10) v[0] = v[10];
			if (threadIdx.x == 11) v[0] = v[11];
			if (threadIdx.x == 12) v[0] = v[12];
			if (threadIdx.x == 13) v[0] = v[13];
			if (threadIdx.x == 14) v[0] = v[14];
			if (threadIdx.x == 15) v[0] = v[15];

			#if (vectorsPerLoop == 32)
			if (threadIdx.x == 16) v[0] = v[16];
			if (threadIdx.x == 17) v[0] = v[17];
			if (threadIdx.x == 18) v[0] = v[18];
			if (threadIdx.x == 19) v[0] = v[19];
			if (threadIdx.x == 20) v[0] = v[20];
			if (threadIdx.x == 21) v[0] = v[21];
			if (threadIdx.x == 22) v[0] = v[22];
			if (threadIdx.x == 23) v[0] = v[23];
			if (threadIdx.x == 24) v[0] = v[24];
			if (threadIdx.x == 25) v[0] = v[25];
			if (threadIdx.x == 26) v[0] = v[26];
			if (threadIdx.x == 27) v[0] = v[27];
			if (threadIdx.x == 28) v[0] = v[28];
			if (threadIdx.x == 29) v[0] = v[29];
			if (threadIdx.x == 30) v[0] = v[30];
			if (threadIdx.x == 31) v[0] = v[31];
			#endif
		}

		if (threadIdx.x < vectorsPerLoop)
			g_S[writeOffset] = v[0];
	}
}

template<typename T>
__launch_bounds__(_warpSize * warpsPerBlock, blocksPerSM)
__global__ void
add32_multi(const T *g_V, T *g_S)
{
	// T v[vectorsPerLoop];
	int readOffset = (blockIdx.x  * intsPerVector * vectorsPerBlock)
				   + (threadIdx.y * intsPerVector * vectorsPerWarp)
				   + (threadIdx.x);
	int writeOffset = (blockIdx.x  * vectorsPerBlock)
	                + (threadIdx.y * vectorsPerWarp)
					+ (threadIdx.x);
		#pragma unroll
		for (int loop = 0; loop < loopsPerWarp; loop++, writeOffset += vectorsPerLoop, readOffset += _warpSize*vectorsPerLoop) {
		// for (int i = 0; i < vectorsPerLoop; i++, readOffset += _warpSize) v[i] = g_V[readOffset];
		// This blob of code can be emitted with the printMultiCode() function.
		// Attempting to write the below code with a series of loops causes the kernel
		//   to die in a fire on my machine (Ubuntu 15.10, GTX 970M, CUDA 7.5).
		// I am told the concise approach might be fixed, or even preferable, on CUDA 8.
		#if vectorsPerLoop == -32
			v[0] += __shfl_xor(v[0], 1);
			v[1] += __shfl_xor(v[1], 1);
			v[2] += __shfl_xor(v[2], 1);
			v[3] += __shfl_xor(v[3], 1);
			v[4] += __shfl_xor(v[4], 1);
			v[5] += __shfl_xor(v[5], 1);
			v[6] += __shfl_xor(v[6], 1);
			v[7] += __shfl_xor(v[7], 1);
			v[8] += __shfl_xor(v[8], 1);
			v[9] += __shfl_xor(v[9], 1);
			v[10] += __shfl_xor(v[10], 1);
			v[11] += __shfl_xor(v[11], 1);
			v[12] += __shfl_xor(v[12], 1);
			v[13] += __shfl_xor(v[13], 1);
			v[14] += __shfl_xor(v[14], 1);
			v[15] += __shfl_xor(v[15], 1);
			v[16] += __shfl_xor(v[16], 1);
			v[17] += __shfl_xor(v[17], 1);
			v[18] += __shfl_xor(v[18], 1);
			v[19] += __shfl_xor(v[19], 1);
			v[20] += __shfl_xor(v[20], 1);
			v[21] += __shfl_xor(v[21], 1);
			v[22] += __shfl_xor(v[22], 1);
			v[23] += __shfl_xor(v[23], 1);
			v[24] += __shfl_xor(v[24], 1);
			v[25] += __shfl_xor(v[25], 1);
			v[26] += __shfl_xor(v[26], 1);
			v[27] += __shfl_xor(v[27], 1);
			v[28] += __shfl_xor(v[28], 1);
			v[29] += __shfl_xor(v[29], 1);
			v[30] += __shfl_xor(v[30], 1);
			v[31] += __shfl_xor(v[31], 1);
			if (threadIdx.x & 1) {
				v[0] = v[1];
				v[2] = v[3];
				v[4] = v[5];
				v[6] = v[7];
				v[8] = v[9];
				v[10] = v[11];
				v[12] = v[13];
				v[14] = v[15];
				v[16] = v[17];
				v[18] = v[19];
				v[20] = v[21];
				v[22] = v[23];
				v[24] = v[25];
				v[26] = v[27];
				v[28] = v[29];
				v[30] = v[31];
			}
			v[0] += __shfl_xor(v[0], 2);
			v[2] += __shfl_xor(v[2], 2);
			v[4] += __shfl_xor(v[4], 2);
			v[6] += __shfl_xor(v[6], 2);
			v[8] += __shfl_xor(v[8], 2);
			v[10] += __shfl_xor(v[10], 2);
			v[12] += __shfl_xor(v[12], 2);
			v[14] += __shfl_xor(v[14], 2);
			v[16] += __shfl_xor(v[16], 2);
			v[18] += __shfl_xor(v[18], 2);
			v[20] += __shfl_xor(v[20], 2);
			v[22] += __shfl_xor(v[22], 2);
			v[24] += __shfl_xor(v[24], 2);
			v[26] += __shfl_xor(v[26], 2);
			v[28] += __shfl_xor(v[28], 2);
			v[30] += __shfl_xor(v[30], 2);
			if (threadIdx.x & 2) {
				v[0] = v[2];
				v[4] = v[6];
				v[8] = v[10];
				v[12] = v[14];
				v[16] = v[18];
				v[20] = v[22];
				v[24] = v[26];
				v[28] = v[30];
			}
			v[0] += __shfl_xor(v[0], 4);
			v[4] += __shfl_xor(v[4], 4);
			v[8] += __shfl_xor(v[8], 4);
			v[12] += __shfl_xor(v[12], 4);
			v[16] += __shfl_xor(v[16], 4);
			v[20] += __shfl_xor(v[20], 4);
			v[24] += __shfl_xor(v[24], 4);
			v[28] += __shfl_xor(v[28], 4);
			if (threadIdx.x & 4) {
				v[0] = v[4];
				v[8] = v[12];
				v[16] = v[20];
				v[24] = v[28];
			}
			v[0] += __shfl_xor(v[0], 8);
			v[8] += __shfl_xor(v[8], 8);
			v[16] += __shfl_xor(v[16], 8);
			v[24] += __shfl_xor(v[24], 8);
			if (threadIdx.x & 8) {
				v[0] = v[8];
				v[16] = v[24];
			}
			v[0] += __shfl_xor(v[0], 16);
			v[16] += __shfl_xor(v[16], 16);
			if (threadIdx.x & 16) {
				v[0] = v[16];
			}
		#endif
		#if vectorsPerLoop == 16
			v[0] += __shfl_xor(v[0], 1);
			v[1] += __shfl_xor(v[1], 1);
			v[2] += __shfl_xor(v[2], 1);
			v[3] += __shfl_xor(v[3], 1);
			v[4] += __shfl_xor(v[4], 1);
			v[5] += __shfl_xor(v[5], 1);
			v[6] += __shfl_xor(v[6], 1);
			v[7] += __shfl_xor(v[7], 1);
			v[8] += __shfl_xor(v[8], 1);
			v[9] += __shfl_xor(v[9], 1);
			v[10] += __shfl_xor(v[10], 1);
			v[11] += __shfl_xor(v[11], 1);
			v[12] += __shfl_xor(v[12], 1);
			v[13] += __shfl_xor(v[13], 1);
			v[14] += __shfl_xor(v[14], 1);
			v[15] += __shfl_xor(v[15], 1);
			if (threadIdx.x & 1) {
			    v[0] = v[1];
			    v[2] = v[3];
			    v[4] = v[5];
			    v[6] = v[7];
			    v[8] = v[9];
			    v[10] = v[11];
			    v[12] = v[13];
			    v[14] = v[15];
			}
			v[0] += __shfl_xor(v[0], 2);
			v[2] += __shfl_xor(v[2], 2);
			v[4] += __shfl_xor(v[4], 2);
			v[6] += __shfl_xor(v[6], 2);
			v[8] += __shfl_xor(v[8], 2);
			v[10] += __shfl_xor(v[10], 2);
			v[12] += __shfl_xor(v[12], 2);
			v[14] += __shfl_xor(v[14], 2);
			if (threadIdx.x & 2) {
			    v[0] = v[2];
			    v[4] = v[6];
			    v[8] = v[10];
			    v[12] = v[14];
			}
			v[0] += __shfl_xor(v[0], 4);
			v[4] += __shfl_xor(v[4], 4);
			v[8] += __shfl_xor(v[8], 4);
			v[12] += __shfl_xor(v[12], 4);
			if (threadIdx.x & 4) {
			    v[0] = v[4];
			    v[8] = v[12];
			}
			v[0] += __shfl_xor(v[0], 8);
			v[8] += __shfl_xor(v[8], 8);
			if (threadIdx.x & 8) {
			    v[0] = v[8];
			}
			v[0] += __shfl_xor(v[0], 16);
		#endif
		// End generated code.

			// ITERATIVE MULTIREDUCTION
			T r[6];
			{
			// 0
			r[0] = g_V[readOffset + 0];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 1
			r[0] = g_V[readOffset + 32];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			r[2] = r[1];
			// 2
			r[0] = g_V[readOffset + 64];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 3
			r[0] = g_V[readOffset + 96];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			if (threadIdx.x & 2) r[2] = r[1];
			r[2] += __shfl_xor(r[2], 4);
			r[3] = r[2];
			// 4
			r[0] = g_V[readOffset + 128];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 5
			r[0] = g_V[readOffset + 160];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			r[2] = r[1];
			// 6
			r[0] = g_V[readOffset + 192];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 7
			r[0] = g_V[readOffset + 224];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			if (threadIdx.x & 2) r[2] = r[1];
			r[2] += __shfl_xor(r[2], 4);
			if (threadIdx.x & 4) r[3] = r[2];
			r[3] += __shfl_xor(r[3], 8);
			r[4] = r[3];
			// 8
			r[0] = g_V[readOffset + 256];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 9
			r[0] = g_V[readOffset + 288];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			r[2] = r[1];
			// 10
			r[0] = g_V[readOffset + 320];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 11
			r[0] = g_V[readOffset + 352];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			if (threadIdx.x & 2) r[2] = r[1];
			r[2] += __shfl_xor(r[2], 4);
			r[3] = r[2];
			// 12
			r[0] = g_V[readOffset + 384];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 13
			r[0] = g_V[readOffset + 416];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			r[2] = r[1];
			// 14
			r[0] = g_V[readOffset + 448];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 15
			r[0] = g_V[readOffset + 480];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			if (threadIdx.x & 2) r[2] = r[1];
			r[2] += __shfl_xor(r[2], 4);
			if (threadIdx.x & 4) r[3] = r[2];
			r[3] += __shfl_xor(r[3], 8);
			if (threadIdx.x & 8) r[4] = r[3];
			r[4] += __shfl_xor(r[4], 16);
			r[5] = r[4];
			// 16
			r[0] = g_V[readOffset + 512];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 17
			r[0] = g_V[readOffset + 544];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			r[2] = r[1];
			// 18
			r[0] = g_V[readOffset + 576];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 19
			r[0] = g_V[readOffset + 608];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			if (threadIdx.x & 2) r[2] = r[1];
			r[2] += __shfl_xor(r[2], 4);
			r[3] = r[2];
			// 20
			r[0] = g_V[readOffset + 640];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 21
			r[0] = g_V[readOffset + 672];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			r[2] = r[1];
			// 22
			r[0] = g_V[readOffset + 704];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 23
			r[0] = g_V[readOffset + 736];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			if (threadIdx.x & 2) r[2] = r[1];
			r[2] += __shfl_xor(r[2], 4);
			if (threadIdx.x & 4) r[3] = r[2];
			r[3] += __shfl_xor(r[3], 8);
			r[4] = r[3];
			// 24
			r[0] = g_V[readOffset + 768];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 25
			r[0] = g_V[readOffset + 800];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			r[2] = r[1];
			// 26
			r[0] = g_V[readOffset + 832];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 27
			r[0] = g_V[readOffset + 864];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			if (threadIdx.x & 2) r[2] = r[1];
			r[2] += __shfl_xor(r[2], 4);
			r[3] = r[2];
			// 28
			r[0] = g_V[readOffset + 896];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 29
			r[0] = g_V[readOffset + 928];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			r[2] = r[1];
			// 30
			r[0] = g_V[readOffset + 960];
			r[0] += __shfl_xor(r[0], 1);
			r[1] = r[0];
			// 31
			r[0] = g_V[readOffset + 992];
			r[0] += __shfl_xor(r[0], 1);
			if (threadIdx.x & 1) r[1] = r[0];
			r[1] += __shfl_xor(r[1], 2);
			if (threadIdx.x & 2) r[2] = r[1];
			r[2] += __shfl_xor(r[2], 4);
			if (threadIdx.x & 4) r[3] = r[2];
			r[3] += __shfl_xor(r[3], 8);
			if (threadIdx.x & 8) r[4] = r[3];
			r[4] += __shfl_xor(r[4], 16);
			if (threadIdx.x & 16) r[5] = r[4];
		}

		if (threadIdx.x < vectorsPerLoop) {
			g_S[writeOffset] = r[5];
			// g_S[writeOffset] = v[0];
		}
	}
}

void printMultiCode(void) {
	for (int k=1; k<vectorsPerLoop; k<<=1) {
		for (int i=0; i<vectorsPerLoop; i+=k) {
			printf("v[%d] += __shfl_xor(v[%d], %d);\n", i,i,k);
		}
		printf("if (threadIdx.x & %d) { \n", k);
		for (int i=0; i<vectorsPerLoop; i+=(k<<1)) printf("    v[%d] = v[%d];\n", i, i+k);
		printf("}\n");
	}
	for (int k=vectorsPerLoop; k<_warpSize; k<<=1) {
		for (int i=0; i<vectorsPerLoop; i+=k) printf("v[%d] += __shfl_xor(v[%d], %d);\n",i,i,k);
	}
}

void printNaiveCode(void) {
	// for (int i = 0; i < vectorsPerLoop; i++)
	// 	printf("v[%d] = g_V[readOffset+%d];\n", i, i*_warpSize);
	for (int j = 1; j < _warpSize; j <<= 1)
		for (int i = 0; i < vectorsPerLoop; i++)
			printf("v[%d] += __shfl_xor(v[%d], %d);\n", i, i, j);

}

template <typename T>
void check(const T * v, const T * s, const int size, const bool debug) {
	int good = 0;
	int bad = 0;
	for (int i=0, j=0; i < size; j++) {
		T t = 0;
		for (int k = 0; k < _warpSize; i++, k++) {
			t += v[i];
		}
		if (s[j] != t) {
			bad++;
			if (debug) {
				if (j < 100) {
					if (j%32==0) printf("\n\n");
					printf("%d: %d %d \n", j, (int)s[j], (int)t);
				}
			}
		} else {
			if (debug && good < 100) printf("                         %d\n", j);
			good++;
		}
	}
	if (bad != 0) printf("Good %d\nBad  %d\n", good, bad);
}

template<typename T>
int test(int argc, char* argv[])
{
	int const device = (argc >= 2) ? atoi(argv[1]) : 0;
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props,device);
	hipSetDevice(device);
	hipError_t err = hipSuccess;

	constexpr int warmups = 16;
	constexpr int runs = 64;
	const int SMs = props.multiProcessorCount;
	const int N = scale * vectorsPerLoop * loopsPerWarp * warpsPerBlock * blocksPerSM * SMs; // number of vectors
	const int k = intsPerVector;
	size_t size = N * k * sizeof(T);
	printf("Total GPU memory usage: %d MB\n", (int)((double)(size + size/k) / (1024*1024) ) );

	T *h_V = (T *)malloc(size);
	T *h_S = (T *)malloc(size/k);
	for (int i = 0; i < N*k; ++i) {
		h_V[i] = (T)(rand() & ((1 << 8)-1)); // Should have enough bits of precision for all types to get exact answers
	}

	T *d_V = NULL;
	T *d_S = NULL;
	hipMalloc((void **)&d_V, size);
	hipMalloc((void **)&d_S, size/k);
	hipMemcpy(d_V, h_V, size, hipMemcpyHostToDevice);

	high_resolution_clock::time_point start, end;
	dim3 threadsPerBlock(_warpSize, warpsPerBlock);
	int blocksPerGrid =(N + vectorsPerBlock - 1) / vectorsPerBlock;

	for (int i=0; i< warmups; i++)
		add32_multi<T><<<blocksPerGrid, threadsPerBlock>>>(d_V, d_S);
	hipDeviceSynchronize();
	start = high_resolution_clock::now();
	for (int i=0; i<runs; i++)
		add32_multi<T><<<blocksPerGrid, threadsPerBlock>>>(d_V, d_S);
	hipDeviceSynchronize();
	end = high_resolution_clock::now();
	double t1 = static_cast<double>(duration_cast<nanoseconds>(end - start).count()) * 1e-9 / static_cast<double>(runs);
	printf("Multireduction (ms):         %.4f\n", t1*1000);
	hipMemcpy(h_S, d_S, size/k, hipMemcpyDeviceToHost);
	check<T>(h_V, h_S, N*k, false);

	for (int i=0; i< warmups; i++)
		add32_naive<T><<<blocksPerGrid, threadsPerBlock>>>(d_V, d_S);
	hipDeviceSynchronize();
	start = high_resolution_clock::now();
	for (int i=0; i<runs; i++)
		add32_naive<T><<<blocksPerGrid, threadsPerBlock>>>(d_V, d_S);
	hipDeviceSynchronize();
	end = high_resolution_clock::now();
	double t2 = static_cast<double>(duration_cast<nanoseconds>(end - start).count()) * 1e-9 / static_cast<double>(runs);
	printf("Previous best practice (ms): %.4f\n", t2*1000);
	printf("Speedup:                     %.4f\n", t2/t1);

	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(h_S, d_S, size/k, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy sums from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	check<T>(h_V, h_S, N*k, false);

	hipFree(d_S);
	hipFree(d_V);
	free(h_S);
	free(h_V);
	hipDeviceReset();
	return 0;
}

int main(int argc, char* argv[]) {
	int const device = (argc >= 2) ? atoi(argv[1]) : 0;
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props,device);
	printf("%s (%2d SMs)\n",props.name, props.multiProcessorCount);
	hipSetDevice(device);

	const int SMs = props.multiProcessorCount;
	const int N = scale * vectorsPerLoop * loopsPerWarp * warpsPerBlock * blocksPerSM * SMs; // number of vectors
	const int k = intsPerVector;
	printf("Using %d vectors of length %d each.\n", N, k);
	printf("Each SM is assigned %d blocks, and can run 8 at once.\n", blocksPerSM);

	int ret = 0;
	printf("\n-- Int\n");
	ret |= test<int>(argc, argv);
	printf("\n-- Float\n");
	ret |= test<float>(argc, argv);
	printf("\n-- Double\n");
	ret |= test<double>(argc, argv);
	return ret;
}
